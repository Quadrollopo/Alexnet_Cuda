// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>




__global__ void matrixMul(int *a, int *b, int *c, int rc, int col) {

    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    if(tx<rc) {
        int x = a[bx*rc+tx] * b[tx*col+by];

        __syncthreads();

        atomicAdd(&c[bx*col+by], x);
    }

}
/**
 *
 * @param m1 first matrix
 * @param m2 second matrix
 * @param rows_1 rows of the first matrix
 * @param col_2 column of the second matrix
 * @param r_c rows of second matrix and columns of first matrix
 * @return result matrix
 * float **m1, float **m2, int rows_1, int col_2, int r_c
 */
void matrix_mul() {
//	rows_1 = 4;
//	col_2 = 2;
//	r_c = 3;
    int a1[12] = {7, 1, 3, 1, 9, 2, 5, 4, 2, 6, 9, 3};
    int b1[6] = {1, 5, 5, 7, 3, 4};
    int c1[8] = {0,0,0,0,0,0,0,0};
    int *d_a, *d_b, *d_c;
    /*cudaMalloc(&d_a, 4 * sizeof(int *));

    for (int i = 0; i < 3; i++) {
        cudaMalloc(&d_a[i], 3 * sizeof(int));
        cudaMemcpy(d_a[i], a1, 3 * sizeof(int), cudaMemcpyHostToDevice);
    }
    cudaMalloc(&d_b, 3 * sizeof(int *));
    for (int i = 0; i < 2; i++) {
        cudaMalloc(&d_b[i], 2 * sizeof(int));
        cudaMemcpy(d_b[i], b1, 6 * sizeof(int), cudaMemcpyHostToDevice);
    }
    cudaMalloc(&d_c, 4 * sizeof(int *));
    for (int i = 0; i < 2; i++) {
        cudaMalloc(&d_c[i], 2 * sizeof(int));
        cudaMemcpy(d_c[i]
                , c1, 8 * sizeof(int), cudaMemcpyHostToDevice);
    }
*/
    hipMalloc(&d_a, 12 * sizeof(int));
    hipMalloc(&d_b, 6 * sizeof(int));
    hipMalloc(&d_c, 8 * sizeof(int));
    hipMemcpy(d_a, a1, 12 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b1, 6 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c1, 8 * sizeof(int), hipMemcpyHostToDevice);

    matrixMul<<<dim3(4,2), 3>>>(d_a,d_b,d_c,3,2);

    hipMemcpy(&c1, d_c, 8 * sizeof(int), hipMemcpyDeviceToHost);
    for(int i=0;i<8;i++)
        printf("%d ",c1[i]);
    hipDeviceReset();
}
/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
//template <int BLOCK_SIZE> __global__ void MatrixMulCUDA(float *C, float *A,
//    float *B, int wA,
//    int wB) {
//  // Block index
//  int bx = blockIdx.x;
//  int by = blockIdx.y;
//
//  // Thread index
//  int tx = threadIdx.x;
//  int ty = threadIdx.y;
//
//  // Index of the first sub-matrix of A processed by the block
//  int aBegin = wA * BLOCK_SIZE * by;
//
//  // Index of the last sub-matrix of A processed by the block
//  int aEnd   = aBegin + wA - 1;
//
//  // Step size used to iterate through the sub-matrices of A
//  int aStep  = BLOCK_SIZE;
//
//  // Index of the first sub-matrix of B processed by the block
//  int bBegin = BLOCK_SIZE * bx;
//
//  // Step size used to iterate through the sub-matrices of B
//  int bStep  = BLOCK_SIZE * wB;
//
//  // Csub is used to store the element of the block sub-matrix
//  // that is computed by the thread
//  float Csub = 0;
//
//  // Loop over all the sub-matrices of A and B
//  // required to compute the block sub-matrix
//  for (int a = aBegin, b = bBegin;
//       a <= aEnd;
//       a += aStep, b += bStep) {
//    // Declaration of the shared memory array As used to
//    // store the sub-matrix of A
//    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
//
//    // Declaration of the shared memory array Bs used to
//    // store the sub-matrix of B
//    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
//
//    // Load the matrices from device memory
//    // to shared memory; each thread loads
//    // one element of each matrix
//    As[ty][tx] = A[a + wA * ty + tx];
//    Bs[ty][tx] = B[b + wB * ty + tx];
//
//    // Synchronize to make sure the matrices are loaded
//    __syncthreads();
//
//    // Multiply the two matrices together;
//    // each thread computes one element
//    // of the block sub-matrix
//#pragma unroll
//
//    for (int k = 0; k < BLOCK_SIZE; ++k) {
//      Csub += As[ty][k] * Bs[k][tx];
//    }
//
//    // Synchronize to make sure that the preceding
//    // computation is done before loading two new
//    // sub-matrices of A and B in the next iteration
//    __syncthreads();
//  }
//
//  // Write the block sub-matrix to device memory;
//  // each thread writes one element
//  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
//  C[c + wB * ty + tx] = Csub;
//}
//
//void ConstantInit(float *data, int size, float val) {
//  for (int i = 0; i < size; ++i) {
//    data[i] = val;
//  }
//}
//
///**
// * Run a simple test of matrix multiplication using CUDA
// */
//int MatrixMultiply(int block_size, const dim3 &dimsA,
//                   const dim3 &dimsB) {
//  // Allocate host memory for matrices A and B
//  unsigned int size_A = dimsA.x * dimsA.y;
//  unsigned int mem_size_A = sizeof(float) * size_A;
//  float *h_A;
//  checkCudaErrors(cudaMallocHost(&h_A, mem_size_A));
//  unsigned int size_B = dimsB.x * dimsB.y;
//  unsigned int mem_size_B = sizeof(float) * size_B;
//  float *h_B;
//  checkCudaErrors(cudaMallocHost(&h_B, mem_size_B));
//  cudaStream_t stream;
//
//  // Initialize host memory
//  const float valB = 0.01f;
//  ConstantInit(h_A, size_A, 1.0f);
//  ConstantInit(h_B, size_B, valB);
//
//  // Allocate device memory
//  float *d_A, *d_B, *d_C;
//
//  // Allocate host matrix C
//  dim3 dimsC(dimsB.x, dimsA.y, 1);
//  unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
//  float *h_C;
//  checkCudaErrors(cudaMallocHost(&h_C, mem_size_C));
//
//  if (h_C == NULL) {
//    fprintf(stderr, "Failed to allocate host matrix C!\n");
//    exit(EXIT_FAILURE);
//  }
//
//  checkCudaErrors(cudaMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
//  checkCudaErrors(cudaMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
//  checkCudaErrors(cudaMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));
//  // Allocate CUDA events that we'll use for timing
//  cudaEvent_t start, stop;
//  checkCudaErrors(cudaEventCreate(&start));
//  checkCudaErrors(cudaEventCreate(&stop));
//
//  checkCudaErrors(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
//
//  // copy host memory to device
//  checkCudaErrors(
//      cudaMemcpyAsync(d_A, h_A, mem_size_A, cudaMemcpyHostToDevice, stream));
//  checkCudaErrors(
//      cudaMemcpyAsync(d_B, h_B, mem_size_B, cudaMemcpyHostToDevice, stream));
//
//  // Setup execution parameters
//  dim3 threads(block_size, block_size);
//  dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);
//
//  // Create and start timer
//  printf("Computing result using CUDA Kernel...\n");
//
//  // Performs warmup operation using matrixMul CUDA kernel
//  if (block_size == 16) {
//    MatrixMulCUDA<16>
//        <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
//  } else {
//    MatrixMulCUDA<32>
//        <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
//  }
//
//  printf("done\n");
//  checkCudaErrors(cudaStreamSynchronize(stream));
//
//  // Record the start event
//  checkCudaErrors(cudaEventRecord(start, stream));
//
//  // Execute the kernel
//  int nIter = 300;
//
//  for (int j = 0; j < nIter; j++) {
//    if (block_size == 16) {
//      MatrixMulCUDA<16>
//          <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
//    } else {
//      MatrixMulCUDA<32>
//          <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
//    }
//  }
//
//  // Record the stop event
//  checkCudaErrors(cudaEventRecord(stop, stream));
//
//  // Wait for the stop event to complete
//  checkCudaErrors(cudaEventSynchronize(stop));
//
//  float msecTotal = 0.0f;
//  checkCudaErrors(cudaEventElapsedTime(&msecTotal, start, stop));
//
//  // Compute and print the performance
//  float msecPerMatrixMul = msecTotal / nIter;
//  double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) *
//                             static_cast<double>(dimsA.y) *
//                             static_cast<double>(dimsB.x);
//  double gigaFlops =
//      (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
//  printf(
//      "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,"
//      " WorkgroupSize= %u threads/block\n",
//      gigaFlops, msecPerMatrixMul, flopsPerMatrixMul, threads.x * threads.y);
//
//  // Copy result from device to host
//  checkCudaErrors(
//      cudaMemcpyAsync(h_C, d_C, mem_size_C, cudaMemcpyDeviceToHost, stream));
//  checkCudaErrors(cudaStreamSynchronize(stream));
//
//  printf("Checking computed result for correctness: ");
//  bool correct = true;
//
//  // test relative error by the formula
//  //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
//  double eps = 1.e-6;  // machine zero
//
//  for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
//    double abs_err = fabs(h_C[i] - (dimsA.x * valB));
//    double dot_length = dimsA.x;
//    double abs_val = fabs(h_C[i]);
//    double rel_err = abs_err / abs_val / dot_length;
//
//    if (rel_err > eps) {
//      printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
//             i, h_C[i], dimsA.x * valB, eps);
//      correct = false;
//    }
//  }
//
//  printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");
//
//  // Clean up memory
//  checkCudaErrors(cudaFreeHost(h_A));
//  checkCudaErrors(cudaFreeHost(h_B));
//  checkCudaErrors(cudaFreeHost(h_C));
//  checkCudaErrors(cudaFree(d_A));
//  checkCudaErrors(cudaFree(d_B));
//  checkCudaErrors(cudaFree(d_C));
//  checkCudaErrors(cudaEventDestroy(start));
//  checkCudaErrors(cudaEventDestroy(stop));
//  printf(
//      "\nNOTE: The CUDA Samples are not meant for performance "
//      "measurements. Results may vary when GPU Boost is enabled.\n");
//
//  if (correct) {
//    return EXIT_SUCCESS;
//  } else {
//    return EXIT_FAILURE;
//  }
//}
//
