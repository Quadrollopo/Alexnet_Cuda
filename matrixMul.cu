#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include "matrixMul.cuh"
#include <hip/hip_runtime.h>




__global__ void matrixMul(float *a, float *b, float *c, int b_row, int b_col) {

    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    if(tx<b_row) {
        int x = a[bx*b_row+tx] * b[tx*b_col+by];

        __syncthreads();

        atomicAdd(&c[bx*b_col+by], x);
    }

}
/**
 * @param values first matrix (1 x weights_row)
 * @param weights second matrix (weights_row x weights_col as array)
 * @param res results matrix (1 x weights_col)
 * @param weights_row rows of the second matrix
 * @param weights_col column of the second matrix
 * float *values, float *weights, float *res, int weights_row, int weights_col
 */
void matrix_mul(float *values, float *weights, float *res, int weights_row, int weights_col) {

    float *d_values, *d_weights, *d_res;


    hipMalloc(&d_values, weights_row * sizeof(float));
    hipMalloc(&d_weights, weights_row * weights_col * sizeof(float));
    hipMalloc(&d_res, weights_col * sizeof(float));

    hipMemcpy(d_values, values, weights_row * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights, weights_row * weights_col * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_res, res, weights_col * sizeof(float), hipMemcpyHostToDevice);

    matrixMul<<<dim3(1,weights_col), weights_row>>>(d_values,d_weights,d_res,weights_row,weights_col);

    hipMemcpy(&res, d_res, weights_col * sizeof(float), hipMemcpyDeviceToHost);

    for(int i=0;i<weights_col;i++)
        printf("%f ",res[i]);

    hipDeviceReset();
}