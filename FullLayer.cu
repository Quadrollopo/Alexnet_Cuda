#include "FullLayer.cuh"

FullLayer::FullLayer(int n_neurons, int linked_neurons, Act func) : Layer(func){
    /**
     * weights: number of rows = num_back_neurons, number of columns = num_neurons
     **/
    this->num_neurons = n_neurons;
    this->num_back_neurons = linked_neurons;
    this->num_weights = n_neurons*linked_neurons;
    std::random_device generator;
    std::uniform_real_distribution<float> weights_rand = std::uniform_real_distribution<float>(0.0f, 0.1f);
    float * tmp_weights = new float[num_weights];
    for (int i=0; i<num_weights; i++){
        tmp_weights[i] = weights_rand(generator);
    }
    hipMalloc(&this->weights,num_weights * sizeof(float));
    hipMemcpy(this->weights, tmp_weights, num_weights * sizeof(float), hipMemcpyHostToDevice);
    delete[] tmp_weights;
    float *tmp_weights_der=new float[num_weights]();
    float *tmp_bias=new float[n_neurons]();
    float *tmp_bias_der=new float[n_neurons]();
    hipMalloc(&this->weights_derivative,num_weights * sizeof(float));
    hipMemset(this->weights_derivative,0,num_weights * sizeof(float));
    hipMalloc(&this->activations,n_neurons * sizeof(float));
    hipMalloc(&this->bias,n_neurons * sizeof(float));
    hipMemset(this->bias,0,n_neurons * sizeof(float));
    hipMalloc(&this->bias_derivative,n_neurons * sizeof(float));
    hipMemset(this->bias_derivative,0,n_neurons * sizeof(float));
    hipMalloc(&this->neurons,n_neurons * sizeof(float));
    hipMalloc(&this->current_weights_derivative,num_weights * sizeof(float));
    hipMalloc(&this->prev_layer_derivative,num_back_neurons * sizeof(float));
    hipMalloc(&this->activation_derivative,num_neurons * sizeof(float));
    hipMalloc(&this->current_bias_derivative,num_neurons * sizeof(float));

    delete[] tmp_weights_der;
    delete[] tmp_bias_der;
    delete[] tmp_bias;
}

FullLayer::~FullLayer(){
    Layer::~Layer();
    hipFree(this->bias);
    hipFree(this->bias_derivative);
    hipFree(this->weights);
    hipFree(this->activations);
    hipFree(this->neurons);
    hipFree(this->weights_derivative);
    hipFree(this->current_weights_derivative);
    hipFree(this->prev_layer_derivative);
    hipFree(this->activation_derivative);
    hipFree(this->current_bias_derivative);
}

float* FullLayer::forward(float *values) {
    matrix_mul(values,
               this->weights,
               activations,
               1,
               this->getNumBackNeurons(),
               this->getNeurons());
    vector_sum(activations,bias,getNeurons());
    activation_func(activations, num_neurons);
    return activations;
}

float* FullLayer::backpropagation(float* cost, float* back_neurons) {
    // other derivatives are obtained in the same way as the bias derivative but using more terms
    // so we start computing bias derivatives and then use those as baseline for other derivatives


    derivative_func(activations, activation_derivative, getNeurons());
    vector_mul(activation_derivative,cost,current_bias_derivative,num_neurons);
    vector_sum(bias_derivative,current_bias_derivative,getNeurons());
    matrix_mul3(back_neurons,
               current_bias_derivative,
               current_weights_derivative,
               this->getNumBackNeurons(),
               1,
               this->getNeurons());
    matrix_mul3(this->weights,
               current_bias_derivative,
               prev_layer_derivative,
               this->getNumBackNeurons(),
               this->getNeurons(),
               1);
    vector_sum(this->weights_derivative,current_weights_derivative,num_weights);
    return prev_layer_derivative;
}

void FullLayer::applyGradient(float lr) {

    vector_constant_mul(this->weights_derivative,lr,num_weights);
    vector_diff(weights,weights_derivative,num_weights);
    vector_constant_mul(bias_derivative,lr,num_neurons);
    vector_diff(bias,bias_derivative,num_neurons);
    hipMemset(this->weights_derivative,0,num_weights * sizeof(float));
    hipMemset(this->bias_derivative,0,num_neurons * sizeof(float));
}

int FullLayer::getNeurons() {
    return num_neurons;
}

int FullLayer::getNumBackNeurons() {
    return num_back_neurons;
}

