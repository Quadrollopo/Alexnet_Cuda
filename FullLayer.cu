#include "hip/hip_runtime.h"
#include "FullLayer.h"

#include <stdexcept>

FullLayer::FullLayer(int n_neurons, int linked_neurons) {
    /**
     * weights: number of rows = weights_len, number of columns = num_neurons
     **/
	this->num_neurons = n_neurons;
	this->weights_len = linked_neurons;
	this->num_weights = n_neurons*linked_neurons;
	this->weights = new float[num_weights];
	this->weights_derivative = new float[num_weights];
    this->activations = new float[n_neurons];
	std::random_device generator;
	std::normal_distribution<float> weights_rand = std::normal_distribution<float>(0.0f, 0.1f);
	for (int i=0; i<n_neurons*linked_neurons; i++){
		weights[i] = weights_rand(generator);
		weights_derivative[i] = 0.0f;
	}
    this->bias = new float[n_neurons];
    this->bias_derivative = new float[n_neurons];
	for (int i=0; i<n_neurons; i++) {
		bias[i] = 0.0f;
		bias_derivative[i] = 0.0f;
	}
}

FullLayer::~FullLayer(){
	delete[] this->weights;
	delete[] this->bias;
    delete[] this->activations;
    delete[] this->weights_derivative;
    delete[] this->bias_derivative;
}

float FullLayer::reLU(float f){
	return f > 0.0f ? f : 0.0f;
}

float sigmoid(float f){
	return 1.f/ (1.f + exp(-f));
}

float der_sigmoid(float f){
	return f*(1 - f);
}

float* FullLayer::forward(float *values) {
	float *val =matrix_mul(values, this->weights, 1, this->weights_len, this->num_neurons);
	//bias sum
	for(int i=0; i<num_neurons; i++){
		val[i] += bias[i];
		val[i] = reLU(val[i]);
        this->activations[i] = val[i];
	}
	return val;
}

int FullLayer::getNeurons() {
    return this->num_neurons;
}

float* FullLayer::getActivations() {
    return this->activations;
}

float FullLayer::Heaviside(float f){
    return f > 0.0f ? 1.0f : 0.0f;
}


float* FullLayer::backpropagation(float* cost, float* back_neurons) {
    // other derivatives are obtained in the same way as the bias derivative but using more terms
    // so we start computing bias derivatives and then use those as baseline for other derivatives
	float* tmp_bias = new float[this->num_neurons];
    for(int i = 0; i < this->num_neurons; i++){
//		tmp_bias[i] = der_sigmoid(this->activations[i])*cost[i];
        tmp_bias[i] = Heaviside(this->activations[i])*cost[i];
		bias_derivative[i] += tmp_bias[i];
    }
	delete[] cost;
//    float* res = matrix_mul(back_neurons, tmp_bias, this->weights_len, 1, this->num_neurons);
	float* res = matrix_mulCPU(back_neurons, tmp_bias, this->weights_len, 1, this->num_neurons);
    float* prev_layer_derivative = matrix_mulCPU(tmp_bias, this->weights, 1, this->num_neurons, this->weights_len);

	delete[] tmp_bias;
	//TODO: Da fare in CUDA
	for (int i=0; i<num_weights; i++){
		weights_derivative[i] += res[i];
	}
	delete[] res;

	return prev_layer_derivative;
}

void FullLayer::applyGradient(float lr) {
	for (int i=0; i<num_weights; i++){
		weights[i] -= weights_derivative[i] * lr;
		weights_derivative[i] = 0;
	}
	for(int i = 0; i < this->num_neurons; i++){
		bias[i] -= bias_derivative[i] * lr;
		bias_derivative[i] = 0;
	}
}
