#include "FullLayer.cuh"

#if CUDA

FullLayer::FullLayer(int n_neurons, int linked_neurons, Act func) : Layer(func){
    /**
     * weights: number of rows = num_back_neurons, number of columns = num_neurons
     **/
    this->num_neurons = n_neurons;
    this->num_back_neurons = linked_neurons;
    this->num_weights = n_neurons*linked_neurons;
    std::random_device generator;
    std::uniform_real_distribution<float> weights_rand = std::uniform_real_distribution<float>(0.0f, 1.0f);
    float * tmp_weights = new float[num_weights];
    for (int i=0; i<num_weights; i++){
        //tmp_weights[i] = weights_rand(generator);
		tmp_weights[i] = 1.0f;
    }
    hipMalloc(&this->weights,num_weights * sizeof(float));
    hipMemcpy(this->weights, tmp_weights, num_weights * sizeof(float), hipMemcpyHostToDevice);
    delete[] tmp_weights;
    float *tmp_weights_der=new float[num_weights]();
    float *tmp_bias=new float[n_neurons]();
    float *tmp_bias_der=new float[n_neurons]();
    hipMalloc(&this->weights_derivative,num_weights * sizeof(float));
    hipMemset(this->weights_derivative,0,num_weights * sizeof(float));
    hipMalloc(&this->activations,n_neurons * sizeof(float));
    hipMalloc(&this->bias,n_neurons * sizeof(float));
    hipMemset(this->bias,0,n_neurons * sizeof(float));
    hipMalloc(&this->bias_derivative,n_neurons * sizeof(float));
    hipMemset(this->bias_derivative,0,n_neurons * sizeof(float));
    hipMalloc(&this->neurons,n_neurons * sizeof(float));

    delete[] tmp_weights_der;
    delete[] tmp_bias_der;
    delete[] tmp_bias;
}

FullLayer::~FullLayer(){
    Layer::~Layer();
    hipFree(this->bias);
    hipFree(this->bias_derivative);
    hipFree(this->weights);
    hipFree(this->activations);
    hipFree(this->neurons);
    hipFree(this->weights_derivative);
}

float* FullLayer::forward(float *values) {
    float *activationss = matrix_mul(values,
                           this->weights,
                           1,
                           this->getNumBackNeurons(),
                           this->getNeurons());

    vector_sum(activationss,bias,getNeurons());
    //hipMemcpy(this->neurons, activationss, this->getNeurons()*sizeof(float), hipMemcpyDeviceToDevice);
    activation_func(activationss, getNeurons());
    hipMemcpy(this->activations, activationss, this->getNeurons()*sizeof(float), hipMemcpyDeviceToDevice);

    return activationss;
}

float* FullLayer::backpropagation(float* cost, float* back_neurons) {
    // other derivatives are obtained in the same way as the bias derivative but using more terms
    // so we start computing bias derivatives and then use those as baseline for other derivatives


    float *der_fun = derivative_func(activations, getNeurons());
    float *current_bias_derivative = vector_mul(der_fun,cost,num_neurons);
    //hipFree(der_fun);
    vector_sum(bias_derivative,current_bias_derivative,getNeurons());
    //hipFree(cost);
    float* current_weights_derivative = matrix_mul(back_neurons,
                                                   current_bias_derivative,
                                                   this->getNumBackNeurons(),
                                                   1,
                                                   this->getNeurons());
    float* prev_layer_derivative = matrix_mul(this->weights,
                                              current_bias_derivative,
                                              this->getNumBackNeurons(),
                                              this->getNeurons(),
                                              1);

    //hipFree(current_bias_derivative);

    vector_sum(weights_derivative,current_weights_derivative,num_weights);


    //hipFree(current_weights_derivative);
    float *x;
    hipMalloc(&x, getNumBackNeurons() * sizeof(float));
    hipMemcpy(x, prev_layer_derivative, getNumBackNeurons() * sizeof(float), hipMemcpyDeviceToDevice);

    return x;
}

void FullLayer::applyGradient(float lr) {
//    float *ee = new float[num_weights];
//    hipMemcpy(ee,weights,num_weights*sizeof (float),hipMemcpyDeviceToHost);
//    printf("\n weights pre diff:\n");
//    for(int i=0;i<this->num_weights;i++)
//        printf("%f ",ee[i]);
//    float *ff = new float[this->getNeurons()];
//    hipMemcpy(ff,bias,this->getNeurons()*sizeof (float),hipMemcpyDeviceToHost);
//    printf("\n bias pre diff:\n");
//    for(int i=0;i<this->getNeurons();i++)
//        printf("%f ",ff[i]);
//    float *cc = new float[num_weights];
//    hipMemcpy(cc,weights_derivative,num_weights*sizeof (float),hipMemcpyDeviceToHost);
//    printf("\n weights derivative:\n");
//    for(int i=0;i<this->num_weights;i++)
//        printf("%f ",cc[i]);
//    float *dd = new float[num_neurons];
//    hipMemcpy(dd,bias_derivative,num_neurons*sizeof (float),hipMemcpyDeviceToHost);
//    printf("\n bias derivative:\n");
//    for(int i=0;i<this->num_neurons;i++)
//        printf("%f ",dd[i]);

    vector_constant_mul(weights_derivative,lr,num_weights);
    vector_diff(weights,weights_derivative,num_weights);
    vector_constant_mul(bias_derivative,lr,num_neurons);
    vector_diff(bias,bias_derivative,num_neurons);

//
//    float *aa = new float[num_weights];
//    hipMemcpy(aa,weights,num_weights*sizeof (float),hipMemcpyDeviceToHost);
//    printf("\n weights post diff:\n");
//    for(int i=0;i<this->num_weights;i++)
//        printf("%f ",aa[i]);
//
//    float *bb = new float[this->getNeurons()];
//    hipMemcpy(bb,bias,this->getNeurons()*sizeof (float),hipMemcpyDeviceToHost);
//    printf("\n bias post diff:\n");
//    for(int i=0;i<this->getNeurons();i++)
//        printf("%f ",bb[i]);
//
//
//    printf("\n\n");
    hipMemset(this->weights_derivative,0,num_weights * sizeof(float));
    hipMemset(this->bias_derivative,0,num_neurons * sizeof(float));
}

int FullLayer::getNeurons() {
    return num_neurons;
}

int FullLayer::getNumBackNeurons() {
    return num_back_neurons;
}


#else

FullLayer::FullLayer(int n_neurons, int linked_neurons, Act func) : Layer(func){
    /**
     * weights: number of rows = weights_len, number of columns = num_neurons
     **/
	this->num_neurons = n_neurons;
	this->num_back_neurons = linked_neurons;
	this->num_weights = n_neurons*linked_neurons;
	this->weights = new float[num_weights];
	this->weights_derivative = new float[num_weights]();
	this->activations = new float[n_neurons];
	this->bias = new float[n_neurons]();
	this->bias_derivative = new float[n_neurons]();
	std::random_device generator;
	std::uniform_real_distribution<float> weights_rand = std::uniform_real_distribution<float>(0.0f, 1.0f);
	for (int i=0; i<num_weights; i++){
		weights[i] = weights_rand(generator);
//		weights[i] = 1.0f;
	}
}

FullLayer::~FullLayer(){
	Layer::~Layer();
	delete[] this->bias;
    delete[] this->bias_derivative;
	delete[] this->weights;
	delete[] this->activations;
	delete[] this->weights_derivative;
}

float* FullLayer::forward(float *values) {
	float *val =matrix_mul(values,
                           this->weights,
                           1,
                           this->getNumBackNeurons(),
                           this->getNeurons());

	for(int i=0; i<getNeurons(); i++){
		val[i] += bias[i];
		val[i] = activation_func(val[i]);
        this->activations[i] = val[i];
	}
	return val;
}

float* FullLayer::backpropagation(float* cost, float* back_neurons) {
    // other derivatives are obtained in the same way as the bias derivative but using more terms
    // so we start computing bias derivatives and then use those as baseline for other derivatives
	float* current_bias_derivative = new float[num_neurons];
    for(int i = 0; i < num_neurons; i++){
        current_bias_derivative[i] = derivative_func(this->activations[i]) * cost[i];
		bias_derivative[i] += current_bias_derivative[i];
    }

	delete[] cost;
	float* current_weights_derivative = matrix_mul_CPU(back_neurons,
                                                       current_bias_derivative,
                                                       this->num_back_neurons,
                                                       1,
                                                       this->num_neurons);
    float* prev_layer_derivative = matrix_mul_CPU(this->weights,
                                                  current_bias_derivative,
                                                  this->getNumBackNeurons(),
                                                  this->getNeurons(),
                                                  1);

	delete[] current_bias_derivative;

	for (int i=0; i<num_weights; i++){
		weights_derivative[i] += current_weights_derivative[i];
	}
//    vector_sum(weights_derivative, current_weights_derivative, num_weights);
//    weights_derivative_CPU = vector_sum_CPU(weights_derivative, current_weights_derivative, num_weights);

	delete[] current_weights_derivative;
	return prev_layer_derivative;
}

void FullLayer::applyGradient(float lr) {
    //fare in CUDA
	for (int i=0; i<num_weights; i++){
		weights[i] -= weights_derivative[i] * lr;
		weights_derivative[i] = 0;
	}
	for(int i = 0; i < this->getNeurons(); i++){
		bias[i] -= bias_derivative[i] * lr;
		bias_derivative[i] = 0;
	}
}

int FullLayer::getNeurons() {
	return num_neurons;
}

int FullLayer::getNumBackNeurons() {
	return num_back_neurons;
}


#endif

