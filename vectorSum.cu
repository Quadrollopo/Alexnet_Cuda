#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>

// CUDA runtime
#include "vectorSum.cuh."
#include <hip/hip_runtime.h>


__global__ void vectorSum(float *a, float *b, float *c, int len){
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    if (id < len)
        c[id] = a[id] + b[id];

}


float* vector_sum(float *a, float *b, int len){
    float *d_a, *d_b, *d_c;

    auto res = new float[len];

    for(int i=0; i < len; i++)
        res[i] = 0.0f;


    hipMalloc(&d_a, len * sizeof(float));
    hipMalloc(&d_b, len * sizeof(float));
    hipMalloc(&d_c, len * sizeof(float));

    hipMemcpy(d_a, a, len * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, len * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, res, len * sizeof(float), hipMemcpyHostToDevice);

    vectorSum<<<1, len>>>(d_a, d_b, d_c, len);

    hipMemcpy(res, d_c, len * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

//    for(int i=0; i < a_row * b_col; i++){
//        printf("%f ", res[i]);
//    }
//    printf("\n\n\n\n");


    hipDeviceReset();

    return res;
}

float* vector_sumCPU(float *a, float *b, int len){
    auto res = new float[len];
    for (int i=0; i<len; i++){
        res[i]=a[i] + b[i];
    }
    return res;
}