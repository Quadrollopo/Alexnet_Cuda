#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>

// CUDA runtime
#include "vectorSum.cuh."
#include <hip/hip_runtime.h>


__global__ void vector_sum_CUDA(float *a, float *b, int len){
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    if (id < len)
        a[id] += b[id];

}


void vector_sum(float *a, float *b, int len){
    float *d_a, *d_b;


    hipMalloc(&d_a, len * sizeof(float));
    hipMalloc(&d_b, len * sizeof(float));

    hipMemcpy(d_a, a, len * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, len * sizeof(float), hipMemcpyHostToDevice);

    vector_sum_CUDA<<<1, len>>>(d_a, d_b, len);

    hipMemcpy(a, d_a, len * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);

//    for(int i=0; i < a_row * b_col; i++){
//        printf("%f ", res[i]);
//    }
//    printf("\n\n\n\n");


    hipDeviceReset();
}

float* vector_sum_CPU(float *a, float *b, int len){
    auto res = new float[len];
    for (int i=0; i<len; i++){
        res[i]=a[i] + b[i];
    }
    return res;
}