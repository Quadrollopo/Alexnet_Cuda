#include "hip/hip_runtime.h"
#include "convolution.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <limits>

__global__ void max_pooling_CUDA(float *image, float *res, int image_size, int pool_size, int stride,  int channel, int res_dim) {

    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;

    if(tx  < channel) {
        float x;
        float max = -3.40282347e+38;

        int index = bx * stride * image_size + by * stride + tx * image_size * image_size; // indice iniziale

        for(int i = 0; i < pool_size; i++){
            for(int j = 0; j < pool_size; j++){
                x = image[index + i * image_size + j];
                if(x>max)
                    max=x;
            }
        }
        res[bx * res_dim + by + tx * res_dim * res_dim] = max;
    }
}

/**
 * @param image first matrix
 * @param kernel second matrix
 * @param image_size size of image
 * @param kernel_size size of kernel
 * @param stride
 * @param pad
 **/

float* max_pooling(float *image, int image_size, int pool_size, int stride, int channel) {
    if(pool_size % 2 == 0){
        std::cout << "Filter size is not odd" << std::endl;
        return nullptr;
    }
    float *d_image, *d_res;
    int res_dim = (image_size-pool_size)/stride+1;
    float* res = new float[res_dim * res_dim * channel]();


    hipMemcpy(d_image, image, image_size * image_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_res, res, res_dim * res_dim * sizeof(float), hipMemcpyHostToDevice);

    max_pooling_CUDA<<<dim3(res_dim, res_dim),channel>>>(d_image,d_res,image_size,pool_size,stride,channel,res_dim);

    hipMemcpy(res, d_res, res_dim * res_dim * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_image);
    hipFree(d_res);

    printf("convolution GPU:\n");
    for(int i=0; i < res_dim * res_dim; i++){
        printf("%.2f ", res[i]);
    }
    printf("\n\n\n");


    hipDeviceReset();

    return res;
}



/**
 * @param a first matrix (1 x weights_row)
 * @param b second matrix (weights_row x weights_col as array)
 * @param a_row rows of the first matrix
 * @param b_row rows of the second matrix
 * @param b_col column of the second matrix
 * float *values, float *weights, int weights_row, int weights_col
 */

float* max_pooling_CPU(float *image, float *kernel, int kern_size, int img_size, int stride, bool pad) {
    return nullptr;

}
