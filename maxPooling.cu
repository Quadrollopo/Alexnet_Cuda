#include "hip/hip_runtime.h"
#include "convolution.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void max_pooling_CUDA(float *image, float *res, int image_size, int kernel_size, int stride, int pad) {

    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;


}

/**
 * @param image first matrix
 * @param kernel second matrix
 * @param image_size size of image
 * @param kernel_size size of kernel
 * @param stride
 * @param pad
 **/

float* max_pooling(float *image, int image_size, int kernel_size, int stride, int pad) {
    if(kernel_size % 2 == 0){
        std::cout << "Filter size is not odd" << std::endl;
        return nullptr;
    }
    if(pad > (kernel_size-1)/2){
        std::cout << "Pad is too high" << std::endl;
        return nullptr;
    }

    float *d_image, *d_res;
    int res_dim = (image_size-kernel_size+2*pad)/stride+1;
    float* res = new float[res_dim * res_dim];

    for(int i=0; i < res_dim * res_dim; i++)
        res[i] = 0.0f;
    int *kernel ;

    hipMalloc(&d_image, image_size * image_size * sizeof(float));
    hipMalloc(&d_res, res_dim * res_dim * sizeof(float));

    hipMemcpy(d_image, image, image_size * image_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_res, res, res_dim * res_dim * sizeof(float), hipMemcpyHostToDevice);

    max_pooling_CUDA<<<1,1>>>(image,res,image_size,kernel_size,stride,)

    hipMemcpy(res, d_res, res_dim * res_dim * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_image);
    hipFree(d_res);

    printf("convolution GPU:\n");
    for(int i=0; i < res_dim * res_dim; i++){
        printf("%.2f ", res[i]);
    }
    printf("\n\n\n");


    hipDeviceReset();

    return res;
}



/**
 * @param a first matrix (1 x weights_row)
 * @param b second matrix (weights_row x weights_col as array)
 * @param a_row rows of the first matrix
 * @param b_row rows of the second matrix
 * @param b_col column of the second matrix
 * float *values, float *weights, int weights_row, int weights_col
 */

float* max_pooling_CPU(float *image, float *kernel, int kern_size, int img_size, int stride, bool pad) {
    return nullptr;

}
