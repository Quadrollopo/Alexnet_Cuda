#include "hip/hip_runtime.h"
#include <iostream>
#include "network.cuh"
#include "CUDA/convolution.cuh"
#include <cmath>
#include <random>
#include <chrono>
#include <memory>
#include <fstream>

using namespace std;

#define BATCH_SIZE 64
#define NUM_EPOCHS 1000
#define NUM_TEST 400
#define IMG_LOAD 60000
#define DECREASE_STEP 600

vector<vector<float>> read_mnist();
vector<uint8_t> read_label();

int main() {
	vector<vector<float>> numbers = read_mnist();
	cout << "numbers loaded" << endl;
	vector<uint8_t> labels = read_label();
	cout << "label loaded" << endl;

	Network net(28, 1,5e-2);
	net.addConvLayer(7, 16, 3, false, reLu)->
	addPoolLayer(2, 2)->
	addConvLayer(3, 32, 1, true, reLu)->
//	addConvLayer(3, 32, 1, true, reLu)->
	addPoolLayer(2, 2)->
	addFullLayer(256, reLu)->
	addFullLayer(256, reLu)->
	addFullLayer(10, softmax);

	random_device r;
	uniform_int_distribution<int> generator = uniform_int_distribution<int>(0 , IMG_LOAD/NUM_TEST - 1);
	float *out, *sol_dev, *numbers_dev;
	float* sol = new float [10]();
	float* out_h = new float [10]();
	hipMalloc(&sol_dev, 10 * sizeof(float));
	hipMalloc(&numbers_dev, numbers[0].size() * sizeof(float));
	vector<int> test_index = vector<int>(NUM_TEST);
	for(int i=0; i<NUM_TEST; i++){
		test_index[i] = generator(r) + i * (IMG_LOAD/NUM_TEST);
	}

	float loss;
	int x = 0;
	ofstream hist_file("./history.txt");
	ofstream val_file("./val.txt");
	for (int j=0; j < NUM_EPOCHS; j++) {
		loss = 0.0;
		for (int i = 0; i < BATCH_SIZE; i++, x++) {
			x = x % IMG_LOAD;
			sol[labels[x]] = 1;
			hipMemcpy(numbers_dev, numbers[x].data(), numbers[x].size(), hipMemcpyHostToDevice);
//			print_CUDA(numbers_dev, numbers[x].size());
			out = net.forward(numbers_dev);
			hipMemcpy(sol_dev, sol, 10 * sizeof(float), hipMemcpyHostToDevice);
			net.train(out, sol_dev, numbers_dev);
			hipMemcpy(out_h, out, 10 * sizeof(float), hipMemcpyDeviceToHost);
			for(int z=0; z < 10; z++)
				loss += -((sol[z] * log(out_h[z] + 1e-10f)) + (1-sol[z]) * log(1-out_h[z] + 1e-10f));

			sol[labels[x]] = 0;
		}
		net.learn();
		loss /= 10 * BATCH_SIZE;
		cout << "loss: " << loss << endl;
		hist_file << loss << " ";
		if(j % 10 == 9) {
			int hit = 0;
			for (int i = 0; i < NUM_TEST; i++) {
				int z = test_index[i];
				hipMemcpy(numbers_dev, numbers[z].data(), numbers[z].size(), hipMemcpyHostToDevice);
				out = net.forward(numbers_dev);
				hipMemcpy(out_h, out, 10 * sizeof(float), hipMemcpyDeviceToHost);

				float max_ix = out_h[0];
				int mx = 0;
				for (int m = 1; m < 10; m++) {
					if (out_h[m] > max_ix) {
						max_ix = out_h[m];
						mx = m;
					}
				}
				if (mx == labels[z])
					hit++;
			}
			cout <<"Test: " << (float) hit/ NUM_TEST << endl;
			val_file << (float) hit/ NUM_TEST << " ";
		}
		if(j % DECREASE_STEP == DECREASE_STEP - 1)
			net.decreaseLR();

	}
	hist_file.close();
	return 0;
}

int reverseInt (int i)
{
	unsigned char ch1, ch2, ch3, ch4;
	ch1=i&255;
	ch2=(i>>8)&255;
	ch3=(i>>16)&255;
	ch4=(i>>24)&255;
	return((int)ch1<<24)+((int)ch2<<16)+((int)ch3<<8)+ch4;
}

vector<vector<float>> read_mnist()
{
	ifstream file ("../train-images.idx3-ubyte", ios::binary);
	if (file.is_open())
	{
		int magic_number=0;
		int number_of_images=0;
		int n_rows=0;
		int n_cols=0;
		file.read((char*)&magic_number,sizeof(magic_number));
		magic_number= reverseInt(magic_number);
		file.read((char*)&number_of_images,sizeof(number_of_images));
		number_of_images= reverseInt(number_of_images);
		file.read((char*)&n_rows,sizeof(n_rows));
		n_rows= reverseInt(n_rows);
		file.read((char*)&n_cols,sizeof(n_cols));
		n_cols= reverseInt(n_cols);
		vector<vector<float>> out = vector<vector<float>>(IMG_LOAD, vector<float>(n_rows*n_cols));
		vector<float> val = vector<float>(n_rows*n_cols);
		for(int i=0;i<IMG_LOAD;++i)
		{
			for(int r=0;r<n_rows;++r)
			{
				for(int c=0;c<n_cols;++c)
				{
					unsigned char temp=0;
					file.read((char*)&temp,sizeof(temp));
					val[r*n_rows + c] = temp / 255.f;
				}
			}
			out[i] = val;
		}
		return out;
	}
	exit(1);
}

vector<uint8_t> read_label(){
	ifstream file ("../train-labels.idx1-ubyte", ios::binary);
	if (file.is_open()) {
		int magic_number = 0;
		int number_of_labels = 0;
		file.read((char *) &magic_number, sizeof(magic_number));
		magic_number = reverseInt(magic_number);
		file.read((char *) &number_of_labels, sizeof(number_of_labels));
		number_of_labels = reverseInt(number_of_labels);
		vector<uint8_t> labels = vector<uint8_t>(IMG_LOAD);
		for(int i=0; i<IMG_LOAD; i++) {
			file.read((char *) &labels[i], sizeof(uint8_t));
		}
		return labels;
	}
	exit(1);
}
