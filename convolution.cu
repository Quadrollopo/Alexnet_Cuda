#include "hip/hip_runtime.h"
#include "convolution.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void convolution_CUDA(float *image, float *kernel, float *res, int image_size, int kernel_size, int stride, int pad, int res_dim) {

    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    if(tx * kernel_size + ty < kernel_size * kernel_size) {

        //controllare bx e by
        int kernel_left = by * stride - pad;
        int kernel_right = kernel_left + kernel_size - 1;
        int kernel_up = bx * stride - pad;
        int kernel_down = kernel_up + kernel_size - 1;

        float x;

        if((kernel_left < 0 && ty < pad) || //padding a sinistra
        (kernel_right >= image_size && ty >= kernel_size - pad) || //padding a destra
        (kernel_up < 0 && tx < pad) || //padding sopra
        (kernel_down >= image_size && tx >= kernel_size - pad)) //padding sotto
            x = 0.0f;

        else{
            int index = ( kernel_up + (kernel_size - 1)/2 ) * image_size + ( kernel_left + (kernel_size - 1)/2 ); // indice centrale
            int offset =  index + ( tx - (kernel_size - 1)/2) * image_size + ty - (kernel_size - 1)/2; // offset da aggiungere  o sottrarre
            x = image[index+offset] * kernel[tx * kernel_size + ty] / (float)kernel_size;
        }
        __syncthreads(); //??

        atomicAdd(&res[by*res_dim + bx], x);
    }

}
/**
 * @param image first matrix
 * @param kernel second matrix
 * @param image_size size of image
 * @param kernel_size size of kernel
 * @param stride
 * @param pad
 **/
float* convolution(float *image, float *kernel, int image_size, int kernel_size, int stride, int pad) {
    if(kernel_size % 2 == 0){
        std::cout << "Filter size is not odd" << std::endl;
        return nullptr;
    }
    if(pad > (kernel_size-1)/2){
        std::cout << "Pad is too high" << std::endl;
        return nullptr;
    }

    float *d_image, *d_kernel, *d_res;
    auto res_dim = (image_size-kernel_size+2*pad)/stride+1;
    auto res = new float[res_dim * res_dim];

    for(int i=0; i < res_dim * res_dim; i++)
        res[i] = 0.0f;


    hipMalloc(&d_image, image_size * image_size * sizeof(float));
    hipMalloc(&d_kernel, kernel_size * kernel_size * sizeof(float));
    hipMalloc(&d_res, res_dim * res_dim * sizeof(float));

    hipMemcpy(d_image, image, image_size * image_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernel_size * kernel_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_res, res, res_dim * res_dim * sizeof(float), hipMemcpyHostToDevice);

    convolution_CUDA<<<dim3(res_dim, res_dim), kernel_size * kernel_size>>>(d_image, d_kernel, d_res, image_size, kernel_size, stride, pad, res_dim);

    hipMemcpy(res, d_res, res_dim * res_dim * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_image);
    hipFree(d_kernel);
    hipFree(d_res);

//    for(int i=0; i < a_row * b_col; i++){
//        printf("%f ", res[i]);
//    }
//    printf("\n\n\n\n");


    hipDeviceReset();

    return res;
}



/**
 * @param a first matrix (1 x weights_row)
 * @param b second matrix (weights_row x weights_col as array)
 * @param a_row rows of the first matrix
 * @param b_row rows of the second matrix
 * @param b_col column of the second matrix
 * float *values, float *weights, int weights_row, int weights_col
 */
float* convolution_CPU(float *image, float *kernel, int kern_size, int img_size, int stride, int pad) {

	int kern_len = kern_size * kern_size;

	float* res = new float [(img_size - 1)*(img_size - 1)];

	for (int x=0; x < img_size - kern_size + 1; x+=img_size){
		for (int y=0; y < img_size - kern_size + 1; y++) {
			float sum = 0;
			for(int i=0; i<kern_size; i+=kern_size){
				for(int j=0; i<kern_size; j++){
					sum += kernel[i + j] * image[x + y];
				}
			}
		}
	}

    return res;
}
