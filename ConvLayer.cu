#include "ConvLayer.cuh"
#include "CUDA/convolution.cuh"
#include "CUDA/vectorSum.cuh"


ConvLayer::ConvLayer(int input_size, int channels, int kernel_size, int kernel_num,
                     int stride, bool pad, Act func)
        : Layer(func){
    this->input_size = input_size;
    this->channels = channels;
    this->kernel_size = kernel_size;
    this->kernel_num = kernel_num;
    this->stride = stride;
    if(pad)
        this->pad = (kernel_size - 1) / 2;
    else
        this->pad = 0;
    this->output_size = ((input_size-kernel_size+2*pad)/stride+1);
    this->output_len = output_size*output_size*kernel_num;
    this->num_weights = kernel_size*kernel_size*channels*kernel_num;

    std::random_device generator;
    std::uniform_real_distribution<float> weights_rand = std::uniform_real_distribution<float>(0.0f, 0.1f);
    float * tmp_weights = new float[num_weights];
    for (int i=0; i<num_weights; i++){
        tmp_weights[i] = weights_rand(generator);
//		weights[i] = 1.0f;
    }
    hipMalloc(&this->weights,num_weights * sizeof(float));
    hipMemcpy(this->weights, tmp_weights, num_weights * sizeof(float), hipMemcpyHostToDevice);
    delete[] tmp_weights;

    hipMalloc(&this->weights_derivative,num_weights * sizeof(float));
    hipMalloc(&this->current_weights_derivative,num_weights * sizeof(float));
    hipMalloc(&this->prev_layer_derivative,input_size*input_size*channels * sizeof(float));
    hipMemset(&this->weights_derivative,0,num_weights * sizeof(float));
    hipMalloc(&this->activations,output_len * sizeof(float));
    hipMalloc(&this->bias,kernel_num * sizeof(float));
    hipMemset(&this->bias,0,kernel_num * sizeof(float));
    hipMalloc(&this->bias_derivative,kernel_num * sizeof(float));
    hipMemset(&this->bias_derivative,0,kernel_num * sizeof(float));
	hipMalloc(&this->activation_derivative,output_len * sizeof(float));

}

    ConvLayer::~ConvLayer(){
        Layer::~Layer();
        hipFree(this->bias);
        hipFree(this->bias_derivative);
        hipFree(this->weights);
        hipFree(this->activations);
        hipFree(this->weights_derivative);

}

float* ConvLayer::forward(float *image) {
    convolution(image,
               this->weights,
               this->activations,
               this->input_size,
               this->kernel_size,
               this->stride,
               this->pad,
               this->channels,
               this->kernel_num);

	vector_sum(this->activations, bias, output_len);
    activation_func(this->activations, output_len);
    return this->activations;
}

int ConvLayer::getInputSize() {
    return this->input_size;
}
int ConvLayer::getChannel() {
    return this->channels;
}
int ConvLayer::getKernelSize() {
    return this->kernel_size;
}
int ConvLayer::getOutputSize() {
    return this->output_size;
}
int ConvLayer::getOutputChannel() {
    return this->kernel_num;
}

void ConvLayer::applyGradient(float lr) {
	vector_constant_mul(weights_derivative,lr,num_weights);
	vector_diff(weights,weights_derivative,num_weights);
	vector_constant_mul(bias_derivative,lr,kernel_num);
	vector_diff(bias,bias_derivative,kernel_num);
	hipMemset(this->weights_derivative,0,num_weights * sizeof(float));
	hipMemset(this->bias_derivative,0,kernel_num * sizeof(float));
}

float *ConvLayer::backpropagation(float *cost, float *back_img) {
	derivative_func(activations, activation_derivative, output_len);
	vector_mul(activation_derivative, cost, activation_derivative, output_len);
	vector_conv_bias(bias_derivative, activation_derivative, output_size*output_size, kernel_num);
	convolution_weights(back_img,
						activation_derivative,
						current_weights_derivative,
						this->input_size,
						this->output_size,
						this->stride,
						pad,
						channels,
						kernel_num);

	vector_sum(weights_derivative, current_weights_derivative, num_weights);

	convolution_prevlayer_backpropagation(activation_derivative,
			this->weights,
			prev_layer_derivative,
			this->output_size,
			this->kernel_size,
			kernel_size - 1,
			kernel_num,
			channels);

	return prev_layer_derivative;
}

int ConvLayer::getNeurons() {
    return output_len;
}

int ConvLayer::getNumBackNeurons() {
    return input_size*input_size*channels;
}
